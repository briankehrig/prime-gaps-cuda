#include "hip/hip_runtime.h"
#include <array>
#include <getopt.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <chrono>
#include <thread>
#include <vector>
#include <unordered_set>
#include <algorithm>
#include <unistd.h>

using namespace std;

#define count_set_bits_64 __builtin_popcountll

#define uint128_t unsigned __int128

#define WIPE_LINE "\r\033[K"
#define END_OF_RANGE ~0

#define RESULT_LIST_SIZE 65536

#ifndef PROGRESS_UPDATE_BLOCKS
#define PROGRESS_UPDATE_BLOCKS 1
#endif

#ifndef MIN_GAP_SIZE
#define MIN_GAP_SIZE 900 // low enough that it will remind people to set it properly
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 46080000000 // we don't actually need to add the UL as far as I know
#endif

#ifndef USE_SPECIFIC_SIEVING_THREAD_BREAKDOWN
#define USE_SPECIFIC_SIEVING_THREAD_BREAKDOWN 0
#endif

#if USE_SPECIFIC_SIEVING_THREAD_BREAKDOWN
#define SIEVING_DUPLICATED_PRIMES 189 // this number is based on the test1, test2, test3 arrays
#else
#define SIEVING_DUPLICATED_PRIMES 80
#endif

#define WORD_LENGTH 120
#define WORD_SIEVING_LENGTH 120

// TODO: THESE ARRAYS HAVE TO CHANGE BASED ON WORD_LENGTH!!

// lower numbers are the least significant digit
__constant__ uint8_t SIEVE_POS_TO_VALUE[32] = {
    1,7,11,13,17,19,23,29,
    31,37,41,43,47,49,53,59,
    61,67,71,73,77,79,83,89,
    91,97,101,103,107,109,113,119,
};

__constant__ uint8_t SIEVE_VALUE_TO_POS[60] = { // WE ARE ONLY TAKING ODD NUMBERS HERE
    0,0,0,1,0,2,3,0,4,5,0,6,0,0,7,
    8,0,0,9,0,10,11,0,12,13,0,14,0,0,15,
    16,0,0,17,0,18,19,0,20,21,0,22,0,0,23,
    24,0,0,25,0,26,27,0,28,29,0,30,0,0,31,
};

__constant__ uint8_t SIEVE_INCREMENTS[8] = {
    6,4,2,4,2,4,6,2,
};

__constant__ uint8_t NEXT_SIEVE_HIT[30] = {
    1,0,5,4,3,2,1,0,3,2,
    1,0,1,0,3,2,1,0,1,0,
    3,2,1,0,5,4,3,2,1,0,
};

__constant__ bool IS_COPRIME_30[15] = {
    1,0,0,1,0,1,1,0,1,1,0,1,0,0,1,
};


__constant__ const uint32_t SHARED_SIZE_WORDS = 12288; // SET THIS TO BE THE TOTAL SIZE OF SHARED MEMORY
__constant__ const uint32_t NUM_SMALL_PRIMES = 23;
__constant__ const uint32_t NUM_MEDIUM_PRIMES = 512 - SIEVING_DUPLICATED_PRIMES;

struct PrimeGap {
    uint128_t startPrime;
    uint32_t gap;
};
bool compareByPrime(const PrimeGap &a, const PrimeGap &b) {
    return a.startPrime < b.startPrime;
}


__device__ uint32_t getSmallMask(uint32_t prime, uint64_t wordOffset) {
    uint32_t word = 0;
    for (int idx=0; idx<32; idx++) {
        if ((prime - (SIEVE_POS_TO_VALUE[idx] % prime)) % prime == wordOffset % prime) {
            word |= 1 << idx;
        }
    }
    if (wordOffset == 0) {
        // For some reason, if we don't have any print statements in this function,
        // it & the makeSmallPrimeWheels function get completely optimized out, and it doesn't modify the
        // 4 wheels at all. 
        // TODO: What variable do I have to mark as volatile so I don't need this hacky code?
        printf("%d ", word % 1000);
    }
    return word;
}


__global__ void makeSmallPrimeWheels(uint32_t* wheel1, uint32_t* wheel2, uint32_t* wheel3, uint32_t* wheel4) {
    uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    if (tidx == 0) {
        printf("Don't optimize everything out lol ");
    }
    for (uint64_t i=tidx; i<7*11*13*17*19*23*29; i+=stride) {
        uint64_t wordStart = i * WORD_LENGTH;
        wheel1[i] |= getSmallMask(7, wordStart);
        wheel1[i] |= getSmallMask(11, wordStart);
        wheel1[i] |= getSmallMask(13, wordStart);
        wheel1[i] |= getSmallMask(17, wordStart);
        wheel1[i] |= getSmallMask(19, wordStart);
        wheel1[i] |= getSmallMask(23, wordStart);
        wheel1[i] |= getSmallMask(29, wordStart);
    }
    
    for (uint64_t i=tidx; i<31*37*41*43*47; i+=stride) {
        uint64_t wordStart = i * WORD_LENGTH;
        wheel2[i] |= getSmallMask(31, wordStart);
        wheel2[i] |= getSmallMask(37, wordStart);
        wheel2[i] |= getSmallMask(41, wordStart);
        wheel2[i] |= getSmallMask(43, wordStart);
        wheel2[i] |= getSmallMask(47, wordStart);
    }
    
    for (uint64_t i=tidx; i<53*59*61*67; i+=stride) {
        uint64_t wordStart = i * WORD_LENGTH;
        wheel3[i] |= getSmallMask(53, wordStart);
        wheel3[i] |= getSmallMask(59, wordStart);
        wheel3[i] |= getSmallMask(61, wordStart);
        wheel3[i] |= getSmallMask(67, wordStart);
    }
    
    for (uint64_t i=tidx; i<71*73*79*83; i+=stride) {
        uint64_t wordStart = i * WORD_LENGTH;
        wheel4[i] |= getSmallMask(71, wordStart);
        wheel4[i] |= getSmallMask(73, wordStart);
        wheel4[i] |= getSmallMask(79, wordStart);
        wheel4[i] |= getSmallMask(83, wordStart);
    }
    if (tidx == 0) {
        printf("\n");
    }
}


__device__ int getBigNumStr(uint128_t result, char* digits) {
    // buffer must be at size 40
    uint32_t digNum = 1;
    for (; (digNum<=40) && result; digNum++) {
        digits[40-digNum] = (char) ('0' + (result % ((uint128_t) 10)));
        result /= 10;
    }
    return 41-digNum;
}


__device__ void clearSieve(uint32_t* sieve, uint32_t sieveLengthWords, uint32_t tidx, uint32_t stride) {
    for (int idx=tidx; idx<sieveLengthWords; idx+=stride) {
        sieve[idx] = 0;
    }
}




__device__ uint64_t lo19(uint128_t n) {
    return (uint64_t) (n % 10000000000000000000UL);
}
__device__ uint64_t hi19(uint128_t n) {
    return (uint64_t) (n / 10000000000000000000UL) % 10000000000000000000UL;
}
uint64_t lo19c(uint128_t n) {
    return (uint64_t) (n % 10000000000000000000UL);
}
uint64_t hi19c(uint128_t n) {
    return (uint64_t) (n / 10000000000000000000UL) % 10000000000000000000UL;
}

__device__ void printBigNum(uint128_t num) {
    // works up to 10^38
    printf("%lu%019lu\n", hi19(num), lo19(num));
}

__device__ uint64_t getMagic(uint128_t mod) {
    // !! THIS ONLY WORKS IF mod > 2^64, otherwise we would get overflow!
    return (uint64_t) ((((uint128_t) 0) - 1) / mod);
}

__device__ uint64_t mul_128_64_hi64_inexact(uint128_t a128, uint64_t b64) {
    // Gets the highest 64 bits of a product of 128-bit and 64-bit integers
    // We are ignoring the lower 64 bits of a128, since that will affect the result by at most 1.
    // We will deal with the +1 later.
    return (uint64_t) (((a128 >> 64) * b64) >> 64);
}

__device__ uint128_t fastMod(uint128_t n, uint128_t mod, uint64_t magic) {
    // !!! THIS ONLY WORKS IF mod > 2^64
    // (this is because "magic" needs to fit in a 64-bit int for efficiency)
    uint128_t result = n - mod * mul_128_64_hi64_inexact(n, magic);

    // potential +1 from losing the carry in the mul128*64, and another +1 from the inexactness of the magic num

    return result - ((result >= mod) + (result >= mod*2)) * mod;
}

__device__ bool fermatTest64(uint128_t n) {
    uint128_t result = 1;
    for (int bit=0; bit<64; bit++) {
        if (n & (1 << bit)) {
            result = (result * 2) % n;
        }
        result = (result * result) % n;
    }
    return (result == 1);
}

__device__ uint128_t squareMod84(uint128_t a, uint128_t mod, uint128_t magic) {
    uint128_t ahi = a>>42;
	uint128_t alo = a & 0x3ffffffffffL;
	//return ((((a*ahi) % m) << 42) + a*alo) % m
    return fastMod((fastMod(a*ahi, mod, magic) << 42) + a*alo, mod, magic);
}

__device__ bool fermatTest645_BRIAN(uint128_t n) {
    // n must be < 2^64.5 ~ 26087635650665564424 = 2.6087e19
    uint128_t result = 1;
    uint128_t mod128 = ((uint128_t) -1) % n + 1;
    uint64_t magic = getMagic(n);
    for (int bit=64; bit>=1; bit--) {
        if ((n >> bit) & 1) {
            result *= 2;
            result -= n * (result >= n); // using an if statement here miiiiight be faster?
        }
        bool overflow = result >= (((uint128_t) 1) << 64);
        result = fastMod(result * result, n, magic);
        if (overflow) {
            result += mod128; // we know that mod128 <= n
            result -= n * (result >= n);
        }
    }
    return result == 1;
}

__device__ bool fermatTest84(uint128_t n) {
    // n must be < 2^64.5 ~ 26087635650665564424 = 2.6087e19
    uint128_t result = 1;
    //uint128_t mod128 = ((uint128_t) -1) % n + 1;
    uint64_t magic = getMagic(n);
    for (int bit=84; bit>=1; bit--) {
        if ((n >> bit) & 1) {
            result *= 2;
            result -= n * (result >= n); // using an if statement here miiiiight be faster?
        }
        result = squareMod84(result, n, magic);
    }
    return result == 1;
}

__device__ int64_t mulmod52(uint64_t a, uint64_t b, uint64_t n, double one_over_n) {
   int64_t tmp, ret;
   double x, y;
   x = (int64_t)a;
   y = (int64_t)b;
   tmp = (int64_t)n * (int64_t)(x * y * one_over_n);
   ret = a * b - tmp;
   if (ret < 0) ret += n;
   else if (ret >= (int64_t)n) ret -= n;
   return ret;
}

__device__ bool fermatTest52(uint64_t n) {
    int64_t result = 1;
    double one_over_n = 1.0 / n;
    for (int bit=51; bit>=0; bit--) {
        result = mulmod52(result, result, n, one_over_n);
        if ((n-1) & (1L << bit)) {
            result *= 2;
        }
    }
    return (result == 1);
}


__device__ uint64_t my_getMagic1(uint128_t mod)
{
	// precomputes (2^128 - 1)/ mod    (a 64 bits number)
	// !! THIS ONLY WORKS IF mod > 2^64, otherwise we would get overflow!
	return (uint64_t) ((((uint128_t) 0) - 1) / mod);
}

__device__ uint128_t my_getMagic2(uint128_t mod, uint64_t magic1)
{
	// precomputes 2^96 % mod    (a 65 bits number)
	//
	// this magic helps later to reduce a 128-bit number r to less than 97 bits
	// magic2 = (1 << 96) % mod
	// r =  (r & ((1 << 96) -1)) + (r >> 96) * magic2
	// 
#if 0
	// slow way
	uint128_t t = 1;
	t <<= 96;
	t %= mod;
#else
	// faster way : barrett reduction
	uint128_t t = (uint128_t) 1 << 96;
	uint128_t e = (uint128_t) magic1 << 32;
	uint64_t e_hi = (uint64_t) (e >> 64);
	uint64_t mod_lo = (uint64_t) mod;
	t -= ((uint128_t) mod_lo * e_hi) + ((uint128_t) e_hi << 64);
	t -= t >= mod ? mod : 0;
#endif
	if (t >> 64)
		t += ((uint128_t) 0xfffffffffffffffeull) << 64;
	return t;
}

// input n : a number up to 64 + 8 = 72 bits
// input mod : the modulus withouts top bit (bit 64 is always 1)
// output result, a number less than 68 bits
//
// This code assumes the compiler knows how to optimize in 1 multiplication
// res_128 = (uint128_t)op1_64 * op2_64
// This code assumes the compiler knows how to optimize the 64 bits shift 
// and the constructions of 128 bits.

__device__ uint128_t my_fastModSqr(uint128_t n, uint64_t mod_lo, uint64_t magic1, uint128_t magic2)
{
	uint64_t n_lo = (uint64_t) n;
	uint64_t n_hi = (uint64_t) (n >> 64);	// let assume n_hi is less than 8 bits

	// step 1
	// do the squaring r = n_lo^2 + n_hi^2 + 2 * n_lo * n_hi;
	uint128_t lo = (uint128_t) n_lo * n_lo;	// lo is less than 64+64 = 128 bits
	uint64_t hi = n_hi * n_hi;	// hi is less than 8 + 8 = 16 bits
	uint128_t mid = (uint128_t) n_lo * (n_hi * 2);	// mid is less than 64 + 16 + 1 = 81 bits
	mid += (uint64_t) (lo >> 64);	// mid is less than 81 -> 82 bits
	lo = (uint64_t) lo;	// lo is less than 64 bits 

	// reduce (r & ((1 << 98) -1)) + (r >> 98) * magic2
	// by doing
	// lo += (hi << (98 - 64) + mid >> (98 - 64)) * magic2
	hi = (hi << 32) + (uint64_t) (mid >> 32);	// hi is less than (16 + 32) (81 - 32) -> 50   bits
	mid = (uint32_t) mid;	// mid is less than 32 bits
	uint64_t magic2_lo = (uint64_t) magic2;	// a 64 bit number
	uint64_t magic2_hi = (uint64_t) (magic2 >> 64);	// a bit mask
	lo += (uint128_t) magic2_lo *hi;	// lo is less than 50+64 = 114 bits
	lo += ((uint128_t) (magic2_hi & hi)) << 64;	// lo is less than 64 + 50 = 114 bits
	// 
	mid += (lo >> 64);	// mid is less than (114 - 64) (32) -> 51 bits
	lo = (uint64_t) lo;	// lo is less than 64 bits
	uint128_t res = (mid << 64) + lo;	// res is less than (51 + 64) (64) -> 116 bits

	// barrett approximate reduction, less than 4 extra bits left
	// magic1 is 64 bits, mid is 52 bits
	uint128_t e = (uint128_t) magic1 * mid;	// e is less than  (51 + 64) = 115 bits
	uint64_t e_hi = (uint64_t) (e >> 64);	// e_hi is less than 51 bits
	res -= ((uint128_t) mod_lo * e_hi) + (((uint128_t) e_hi) << 64);
	// - barrett reduction : 1 extra subtraction sometimes needed  (about less 50 %)
	// - barrett magic1 is underestimated by up to 1 : 1 extra subtraction (rarely needed)
	// -> res is less than 3 times the modulus, i.e about 0x6xxx...xxx (67 bit number)

	return res;

}


#if !defined(EULER_CRITERION)
#define EULER_CRITERION 1
#endif
__device__ bool fermatTest65(uint128_t n)
{

	// - iterate on 64 bits before squaring would overflow 
	// and until modular reduction becomes necessary
	// - hardcode 2 most significant bits
	// - Advance 2 bits at a time with window size 2
	uint64_t n_lo = (uint64_t) n;
	uint64_t result64 = ((n_lo >> 63) & 1) ? 8 : 4;	// 2 most significant bits of the modulus 
	int bit = 63;
	while (bit && result64 <= 38967) {
		bit -= 2;
		result64 *= result64;
		result64 *= result64;
		result64 *= 1 << ((n_lo >> bit) & 3);
	}

	// - iterate on 128 bits with modular reduction
	// - at each step and intermediate numbers are only
	//   a few bits larger than the modulus
	// - the last iteration is optimized out the loop
	uint128_t result = result64;
	uint64_t magic1 = my_getMagic1(n);
	uint128_t magic2 = my_getMagic2(n, magic1);
	while (bit > 1) {
		// advance 2 bits at a time
		bit -= 2;
		// square and reduce
		result = my_fastModSqr(result, n, magic1, magic2);
		result = my_fastModSqr(result, n, magic1, magic2);
		// - multiply with window size 2     (values are 1, 2, 4 or 8)
		// and let the number overflow a little bit
		// - result is less than 64 + 3 + 4 = 71 bits   (quite over-rounded)
		result *= 1 << ((n_lo >> bit) & 3);
	}

#if EULER_CRITERION
	// - last round with 1 bit to process
	// - Euler's criterion 2^(n>>1) == legendre_symbol(2,n) (https://en.wikipedia.org/wiki/Euler%27s_criterion)
	// - This skips the modexp last round. Thanks Mr Leonhard.
	// - shortcut calculation of legendre symbol (https://en.wikipedia.org/wiki/Legendre_symbol)
	// legendre_symbol(2,n) = 1 if n = 1 or 7 mod 8     (when bits 1 and 2 are same)
	// legendre_symbol(2,n) = -1 if n = 3 or 5 mod 8    (when bits 1 and 2 are different)
	uint64_t legendre = ((n_lo >> 1) & 1) ^ ((n_lo >> 2) & 1);	// shortcut calculation of legendre symbol
	uint128_t expected = legendre ? n - 1 : 1;	// shortcut calculation of legendre symbol

	// - final reductions :  result %= n;
	// - the last operation was a multiplication by 1,2,4, or 8, without reduction. 
	//   therefore, there are many extra bits to shave. worst case is 7 bits.
	// - use repeated subtractions within a log2 algorithm
	uint128_t t = n;
#if 1
	while (t < result)
		t *= 2;
#else
	t <<= 63 - __builtin_clzll((uint64_t)(result >> 64));
#endif
	while (t >= n) {
		if (result >= t) {
			result -= t;
		}
		t >>= 1;
	}

#else
	// - last round with 1 bit to process , and this bit from n-1 is always 0. No multiplication by 2 is needed
	result = my_fastModSqr(result, n, magic1, magic2);
	uint128_t expected = 1;

	// - final reductions :  result %= n;
	// - only a few bits to shave
	while (result >= n) {
		result -= n;
	}
#endif

	return result == expected;
}


__device__ void sieveSmallPrimes(uint32_t* sieve, uint32_t sieveLengthWords, uint128_t start,
                                 uint32_t* smallPrimeWheel1, uint32_t* smallPrimeWheel2,
                                 uint32_t* smallPrimeWheel3, uint32_t* smallPrimeWheel4) {

    // sieve should be in SHARED MEMORY for this function to work properly
    for (uint32_t i = threadIdx.x; i < sieveLengthWords; i += blockDim.x) {
        // TODO: Check if these modulos are getting optimized, especially for int128

        uint128_t wordStart = start/WORD_LENGTH + i;
        // We cannot replace the atomicOr with a non-atomic operation, because that might skip sieving out some values
        // and we can't miss any because of pseudoprimes
        // making this non-atomic actually doesn't increase performance at all, so the bottleneck is elsewhere on my laptop
        atomicOr(&sieve[i], smallPrimeWheel1[wordStart % (7*11*13*17*19*23*29)] | 
                            smallPrimeWheel2[wordStart % (31*37*41*43*47)] | 
                            smallPrimeWheel3[wordStart % (53*59*61*67)] | 
                            smallPrimeWheel4[wordStart % (71*73*79*83)]);
        

    }
    __syncthreads();
}

__device__ void sieveMediumLargePrimesInner(uint32_t* sieve, uint32_t sieveLengthWords, uint128_t start, uint32_t p) {
    // this function could do with some optimization overall
    uint128_t x = start / p + 1;
    x = x + NEXT_SIEVE_HIT[x % 30]; // the next number k after start/p that's k for gcd(k,30) = 1
    uint32_t pMultSieveIndex = SIEVE_VALUE_TO_POS[(x % 30) / 2];
    x *= p; // the next number p*k after start that has gcd(k,30) = 1
    uint32_t currentWord = (uint32_t) ((x - start) / WORD_LENGTH);
    uint32_t currentPosInWord = x % WORD_LENGTH;
    while (currentWord < sieveLengthWords) {
        // Update the sieve
        if (currentPosInWord < WORD_SIEVING_LENGTH) { // TODO: THIS CHECK LOSES A BUNCH OF TIME
            atomicOr(&sieve[currentWord], 1 << SIEVE_VALUE_TO_POS[currentPosInWord / 2]);
        }

        // Find the next position
        currentPosInWord += p * SIEVE_INCREMENTS[pMultSieveIndex];
        pMultSieveIndex = (pMultSieveIndex + 1) % 8;
        currentWord += (currentPosInWord) / WORD_LENGTH;
        currentPosInWord %= WORD_LENGTH;
    }
}

__device__ uint32_t test1[256] = {11,11,11,11,11,11,11,11,11,11,11,10,10,10,10,10,10,10,10,10,10,9,9,9,9,9,9,9,9,9,9,9,9,9,9,9,9,9,9,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,7,7,7,7,7,7,7,7,7,7,7,7,7,7,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,5,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,3,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2,2};
__device__ uint32_t test2[256] = {0,1,2,3,4,5,6,7,8,9,10,0,1,2,3,4,5,6,7,8,9,0,1,2,3,4,5,6,7,8,0,1,2,3,4,5,6,7,8,0,1,2,3,4,5,6,7,0,1,2,3,4,5,6,7,0,1,2,3,4,5,6,7,0,1,2,3,4,5,6,0,1,2,3,4,5,6,0,1,2,3,4,5,0,1,2,3,4,5,0,1,2,3,4,5,0,1,2,3,4,5,0,1,2,3,4,0,1,2,3,4,0,1,2,3,4,0,1,2,3,4,0,1,2,3,4,0,1,2,3,4,0,1,2,3,0,1,2,3,0,1,2,3,0,1,2,3,0,1,2,3,0,1,2,3,0,1,2,3,0,1,2,3,0,1,2,0,1,2,0,1,2,0,1,2,0,1,2,0,1,2,0,1,2,0,1,2,0,1,2,0,1,2,0,1,2,0,1,2,0,1,2,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1};
__device__ uint32_t test3[256] = {0,0,0,0,0,0,0,0,0,0,0,1,1,1,1,1,1,1,1,1,1,2,2,2,2,2,2,2,2,2,3,3,3,3,3,3,3,3,3,4,4,4,4,4,4,4,4,5,5,5,5,5,5,5,5,6,6,6,6,6,6,6,6,7,7,7,7,7,7,7,8,8,8,8,8,8,8,9,9,9,9,9,9,10,10,10,10,10,10,11,11,11,11,11,11,12,12,12,12,12,12,13,13,13,13,13,14,14,14,14,14,15,15,15,15,15,16,16,16,16,16,17,17,17,17,17,18,18,18,18,18,19,19,19,19,20,20,20,20,21,21,21,21,22,22,22,22,23,23,23,23,24,24,24,24,25,25,25,25,26,26,26,26,27,27,27,28,28,28,29,29,29,30,30,30,31,31,31,32,32,32,33,33,33,34,34,34,35,35,35,36,36,36,37,37,37,38,38,38,39,39,39,40,40,41,41,42,42,43,43,44,44,45,45,46,46,47,47,48,48,49,49,50,50,51,51,52,52,53,53,54,54,55,55,56,56,57,57,58,58,59,59,60,60,61,61,62,62,63,63,64,64,65,65,66,66};
__device__ void sieveMediumPrimes(uint32_t* sieve, uint32_t sieveLengthWords, uint128_t start,
                                  uint32_t* primeList, uint32_t primeCount) {
    // sieve should be in SHARED MEMORY for this function to work properly
    // from the perspective of this function, the first 23 primes DON'T EXIST!!!!!

    /*
    (1): array that replaces the individiual numbers (8,8,8,8....4,4,4,4.....1,1,1,1)
    (2): array that replaces threadIdx.x % 8 (0,1,2,3,4,5,6,7,0,1,2,3.....0,1,0,1,0,1....0,0,0,0)
    (3): array that has the prime indexes: (0,0,0,0,0,0,0,0,1,1,1.....237,238,239)
    starting pos in the sieve: depends on (2)
    length of the search space in sieve: depends on (1)
    starting N: depends on (2)
    prime: depends on (3)
    */
#if USE_SPECIFIC_SIEVING_THREAD_BREAKDOWN
    // This code works better on my laptop GPU
    if (threadIdx.x < 256) {
        uint32_t numBlocks = test1[threadIdx.x];
        uint32_t blockIdx = test2[threadIdx.x];
        uint32_t primeIdx = test3[threadIdx.x];
        uint32_t wordStart = (uint32_t) (((double) sieveLengthWords) * blockIdx / numBlocks);
        uint32_t wordEnd   = (uint32_t) (((double) sieveLengthWords) * (blockIdx+1) / numBlocks);
        // Note: If numBlocks is not a factor of sieveLengthWords, we might end up skipping the last word!!
        sieveMediumLargePrimesInner(
            sieve + wordStart,
            wordEnd - wordStart,
            start + wordStart*WORD_LENGTH,
            primeList[primeIdx]
        );
    }
    for (uint32_t pidx = threadIdx.x+blockDim.x-189; pidx < primeCount; pidx += blockDim.x) {
        sieveMediumLargePrimesInner(sieve, sieveLengthWords, start, primeList[pidx]);
    }
#else
    // This code works better on an RTX 4090
    if (threadIdx.x < 64) {
        sieveMediumLargePrimesInner(sieve + (sieveLengthWords/4 * (threadIdx.x % 4)), sieveLengthWords/4,
            start + (sieveLengthWords * WORD_LENGTH/4 * (threadIdx.x % 4)), primeList[threadIdx.x/4]);
    } else if (threadIdx.x < 128) {
        sieveMediumLargePrimesInner(sieve + (sieveLengthWords/2 * (threadIdx.x % 2)), sieveLengthWords/2,
            start + (sieveLengthWords * WORD_LENGTH/2 * (threadIdx.x % 2)), primeList[(threadIdx.x-64)/2 + 16]);
    } else {
        sieveMediumLargePrimesInner(sieve, sieveLengthWords, start, primeList[threadIdx.x-80]);
    }
    for (uint32_t pidx = threadIdx.x+blockDim.x-80; pidx < primeCount; pidx += blockDim.x) {
        sieveMediumLargePrimesInner(sieve, sieveLengthWords, start, primeList[pidx]);
    }
#endif
    /*
    if (threadIdx.x < 256) {
        if (threadIdx.x < 128) {
            sieveMediumLargePrimesInner(sieve + (sieveLengthWords/8 * (threadIdx.x % 8)), sieveLengthWords/8,
                start + (sieveLengthWords * WORD_LENGTH/8 * (threadIdx.x % 8)), primeList[threadIdx.x/8]);
        } else {
            sieveMediumLargePrimesInner(sieve + (sieveLengthWords/4 * (threadIdx.x % 4)), sieveLengthWords/4,
                start + (sieveLengthWords * WORD_LENGTH/4 * (threadIdx.x % 4)), primeList[(threadIdx.x-128)/2 + 16]);
        }
    } else {
        if (threadIdx.x < 384) {
            sieveMediumLargePrimesInner(sieve + (sieveLengthWords/2 * (threadIdx.x % 2)), sieveLengthWords/2,
                start + (sieveLengthWords * WORD_LENGTH/2 * (threadIdx.x % 2)), primeList[(threadIdx.x-256)/2 + 16+32]);
        } else {
            sieveMediumLargePrimesInner(sieve, sieveLengthWords, start, primeList[threadIdx.x-272]);
        }
    }
    for (uint32_t pidx = threadIdx.x+blockDim.x-272; pidx < primeCount; pidx += blockDim.x) {
        sieveMediumLargePrimesInner(sieve, sieveLengthWords, start, primeList[pidx]);
    }
    */
    
    __syncthreads();
}

__device__ void sieveLargePrimes(uint32_t* sieve, uint32_t sieveLengthWords, uint128_t start,
                                 uint32_t* primeList, uint32_t primeCount, uint32_t numBlocks) {
    // sieve should be in GLOBAL MEMORY for this function to work properly
    
    uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * numBlocks;

    for (uint32_t pidx = tidx; pidx < primeCount; pidx += stride) {
        sieveMediumLargePrimesInner(sieve, sieveLengthWords, start, primeList[pidx]);
    }
    __syncthreads();
}

__device__ void sievePseudoprimes(uint32_t* sieve, uint32_t sieveLengthWords, uint128_t start,
                                  uint32_t* primeList, uint32_t* rhoList, uint32_t primeCount,
                                  uint32_t numBlocks) {
    // sieve should be in GLOBAL MEMORY for this function to work properly

    // We are sieving for entries that are congruent to p mod p*rho(p), because this is guaranteed
    // to remove all 2-PSPs that have p as a factor.
    // If we do this up to p=5000000, then all remaining 2-PSPs (i.e. with no prime factors below 5M)
    //   have been checked up to 2^65, with no gaps of size >900.
    uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * numBlocks;

    for (uint32_t pidx = tidx; pidx < primeCount; pidx += stride) {
        uint32_t p = primeList[pidx];
        uint32_t rho = rhoList[pidx];
        uint64_t pTimesRho = ((uint64_t) p) * rho;
        pTimesRho <<= pTimesRho % 2; // if it's odd, multiply it by 2
        uint64_t position = pTimesRho + p - (start % pTimesRho);
        position -= pTimesRho * (position > pTimesRho);
        
        uint64_t currentWord = position / WORD_LENGTH; // this needs to be 64 bit
        uint32_t currentPosInWord = position % WORD_LENGTH;

        uint32_t pTimesRhoMod120 = pTimesRho % WORD_LENGTH;
        uint64_t pTimesRhoDiv120 = pTimesRho / WORD_LENGTH;
        while (currentWord < sieveLengthWords) {
            // Update the sieve
            if (currentPosInWord < WORD_SIEVING_LENGTH && IS_COPRIME_30[(currentPosInWord % 30) / 2]) {
                /*if ((~sieve[currentWord]) & (1 << SIEVE_VALUE_TO_POS[currentPosInWord / 2])) {
                    uint128_t num = start + ((uint128_t) currentWord)*WORD_LENGTH + currentPosInWord;
                    if (fermatTest645(num)) {
                        printf("Pseudoprime p=%d mod 1e19=%lu %lu %u %u\n", p, (uint64_t) (num % 10000000000000000000UL),
                        currentWord, currentPosInWord, (uint32_t) (num%p));
                        //printf("20000%lu\n", (uint64_t) (num % 10000000000000000000UL));
                    }
                }*/
                atomicOr(&sieve[currentWord], 1 << SIEVE_VALUE_TO_POS[currentPosInWord / 2]);
            }

            // Find the next position
            currentPosInWord += pTimesRhoMod120;
            currentWord += pTimesRhoDiv120 + (currentPosInWord >= WORD_LENGTH);
            currentPosInWord -= WORD_LENGTH * (currentPosInWord >= WORD_LENGTH);
        }
    }
    __syncthreads();
}

__device__ void sieveAll(uint32_t* globalSieve, uint128_t sieveStart, uint32_t sieveLengthWords,
                         uint32_t* primeList, uint32_t* rhoList, uint32_t primeCount,
                         uint32_t* smallPrimeWheel1, uint32_t* smallPrimeWheel2,
                         uint32_t* smallPrimeWheel3, uint32_t* smallPrimeWheel4,
                         uint32_t numBlocks) {
    // the actual sieve length is WORD_LENGTH * sieveLengthWords

    uint32_t tidx = blockIdx.x * numBlocks + threadIdx.x;
    
    if (sieveLengthWords % SHARED_SIZE_WORDS != 0) {
        if (tidx == 0) {
            printf("ERROR: Length of the block (%lu) is not a multiple of %d times the shared size (%d)\n",
                   ((uint64_t) sieveLengthWords)*WORD_LENGTH, WORD_LENGTH, SHARED_SIZE_WORDS);
        }
        return;
    }

    __shared__ uint32_t sharedSieve[SHARED_SIZE_WORDS];
    uint32_t numSharedBlocks = sieveLengthWords / SHARED_SIZE_WORDS;
    
    for (uint64_t sharedBlockIdx = blockIdx.x; sharedBlockIdx < numSharedBlocks; sharedBlockIdx += numBlocks) {
        // Reset the shared memory to 0, since it doesn't necessarily start out that way
        for (int idx=threadIdx.x; idx<SHARED_SIZE_WORDS; idx+=blockDim.x) {
            sharedSieve[idx] = 0;
        }

        sieveSmallPrimes(sharedSieve, SHARED_SIZE_WORDS, sieveStart + sharedBlockIdx*SHARED_SIZE_WORDS*WORD_LENGTH,
                         smallPrimeWheel1, smallPrimeWheel2, smallPrimeWheel3, smallPrimeWheel4);

        if (NUM_MEDIUM_PRIMES > 0) {
            sieveMediumPrimes(sharedSieve, SHARED_SIZE_WORDS, sieveStart + sharedBlockIdx*SHARED_SIZE_WORDS*WORD_LENGTH,
                            primeList+NUM_SMALL_PRIMES, NUM_MEDIUM_PRIMES);
        }

        for (int sharedIdx=threadIdx.x; sharedIdx<SHARED_SIZE_WORDS; sharedIdx += numBlocks) {
            atomicOr(&globalSieve[sharedBlockIdx*SHARED_SIZE_WORDS + sharedIdx], sharedSieve[sharedIdx]);
        }
    }

#if 0
        sieveLargePrimes(globalSieve, sieveLengthWords, sieveStart,
                         primeList+NUM_SMALL_PRIMES+NUM_MEDIUM_PRIMES, primeCount-NUM_SMALL_PRIMES-NUM_MEDIUM_PRIMES,
                         numBlocks);

#endif
    sievePseudoprimes(globalSieve, sieveLengthWords, sieveStart,
                      primeList+NUM_SMALL_PRIMES+NUM_MEDIUM_PRIMES,
                      rhoList+NUM_SMALL_PRIMES+NUM_MEDIUM_PRIMES,
                      primeCount-NUM_SMALL_PRIMES-NUM_MEDIUM_PRIMES, numBlocks);
    
}

__global__ void kernel(uint32_t* globalSieve, uint128_t sieveStart, uint32_t sieveLengthWords,
                       uint32_t* primeList, uint32_t* rhoList, uint32_t primeCount,
                       uint32_t* smallPrimeWheel1, uint32_t* smallPrimeWheel2,
                       uint32_t* smallPrimeWheel3, uint32_t* smallPrimeWheel4) {
    sieveAll(globalSieve, sieveStart, sieveLengthWords, primeList, rhoList, primeCount,
        smallPrimeWheel1, smallPrimeWheel2, smallPrimeWheel3, smallPrimeWheel4, gridDim.x);
}



// ========== THIS IS THE START OF THE 2ND PART OF THE CODE, PARSING THE SIEVE TO FIND PRIME GAPS ========== //


__device__ uint64_t findNextUnsieved(uint32_t* sieve, uint64_t sieveLengthWords, uint64_t bitPosition) {
    // bitPosition treats the entire sieve as a bitSet
    // Gets the next unsieved number from a given position,
    //   starting with AND INCLUDING bitPosition
    
    if (bitPosition >= sieveLengthWords*32) return END_OF_RANGE;
    int64_t wordIdx = bitPosition/32;
    while (wordIdx < sieveLengthWords) { // total number of bits in shared memory
        uint32_t word = ~(sieve[wordIdx]) & ((~0U) << (bitPosition%32));
        if (word) {
            return wordIdx*32 + __ffs(word) - 1;
        }
        wordIdx++;
        bitPosition = 0; // maybe can optimize this more
    }
    return END_OF_RANGE; // return 0xFFFFFFFF if no result
}

__device__ uint64_t findPrevUnsieved(uint32_t* sieve, int64_t bitPosition) {
    // bitPosition treats the entire sieve as a bitSet
    // Gets the previous unsieved number from a given position,
    //   starting with AND INCLUDING bitPosition
    
    if (bitPosition < 0) return END_OF_RANGE;
    int64_t wordIdx = bitPosition/32; // signed int, so we can compare it with 0 properly
    while (wordIdx >= 0) { // total number of bits in shared memory
        uint32_t word = (~sieve[wordIdx]) & ((~0U) >> (31 - bitPosition%32));
        if (word) {
            return wordIdx*32 + 31 - __clz(word);
        }
        wordIdx--;
        bitPosition = 31; // maybe can optimize this more
    }
    return END_OF_RANGE; // return 0xFFFFFFFF if no result
}

__device__ uint128_t getNumberFromSieve(uint128_t start, int64_t bitPosition) {
    return start + bitPosition/32*WORD_LENGTH + SIEVE_POS_TO_VALUE[bitPosition%32];
}

#define FERMAT_TEST fermatTest65
__device__ void findGaps(uint32_t* sieve, uint128_t start, uint64_t sieveLengthWords, uint32_t startBlock, PrimeGap* resultList) {
    // sieve should be in GLOBAL MEMORY for this function to work properly
    uint32_t gridDimNew = gridDim.x - startBlock;
    uint32_t blockIdxNew = blockIdx.x - startBlock;

    const int MIN_GAP_SIZE_30 = MIN_GAP_SIZE / 30;
    // one group of 30 numbers corresponds to 8 bits

    int64_t bitPosition;
    int64_t limitBitPosition;
    uint32_t tidx = blockIdxNew * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDimNew;
    bitPosition = sieveLengthWords*32 / stride * tidx;
    limitBitPosition = sieveLengthWords*32 / stride * (tidx + 1);
    
    bitPosition -= bitPosition % 32;
    limitBitPosition -= limitBitPosition % 32;

    bool hitEndOfRange = false;
    bitPosition = findNextUnsieved(sieve, sieveLengthWords, bitPosition);

    // Calculate the first prime in the range
    uint128_t lastPrime = getNumberFromSieve(start, bitPosition);
    while (!FERMAT_TEST(lastPrime)) {
        if (bitPosition == END_OF_RANGE) {
            // this will only happen if we find a single gap of size sieveLengthWords*WORD_LENGTH/threadsPerBlock
            // which almost certainly will never happen
            // if we ever hit the end of the shared memory range while finding a gap, we can ignore it
            // because we will find that gap anyway while searching on the borders of shared memory blocks
            hitEndOfRange = true;
            break;
        }
        bitPosition = findNextUnsieved(sieve, sieveLengthWords, ++bitPosition);
        lastPrime = getNumberFromSieve(start, bitPosition);
    }
    
    __syncthreads();
    
    bool isPrime = false;
    if (hitEndOfRange) goto endLabel;

    bitPosition += 8 * MIN_GAP_SIZE_30;

    while (true) {
        bitPosition = findPrevUnsieved(sieve, --bitPosition);
        uint128_t toTest = start + bitPosition/32*WORD_LENGTH + SIEVE_POS_TO_VALUE[bitPosition%32];
        if (toTest == lastPrime) {
            // found a large gap! this part will get entered rarely so I don't really have to optimize it
            bitPosition += 8 * MIN_GAP_SIZE_30;
            bitPosition = findNextUnsieved(sieve, sieveLengthWords, bitPosition);
            uint128_t upperPrime = getNumberFromSieve(start, bitPosition);
            while (!FERMAT_TEST(upperPrime)) {
                if (bitPosition == END_OF_RANGE) goto endLabel;
                bitPosition = findNextUnsieved(sieve, sieveLengthWords, ++bitPosition);
                upperPrime = getNumberFromSieve(start, bitPosition);
            }
            uint32_t gap = (uint32_t) (upperPrime - lastPrime);
            
            int resultIdx = atomicAdd(&resultList[0].gap, 1) + 1; // index 0 of the list keeps track of the length
            resultList[resultIdx].startPrime = lastPrime;
            resultList[resultIdx].gap = gap;

            lastPrime = upperPrime;
            bitPosition += 8 * MIN_GAP_SIZE_30;
        } else {
            isPrime = FERMAT_TEST(toTest);
            if (isPrime) lastPrime = toTest;
        }

        if (bitPosition >= limitBitPosition && isPrime) break;
        bitPosition += 8 * MIN_GAP_SIZE_30 * isPrime;
        // for some reason, it doesn't work if I just put this in the while loop condition
    }
    endLabel:
    __syncthreads();
}


__global__ void kernel2(uint32_t* globalSieve, uint128_t sieveStart, uint64_t sieveLengthWords, PrimeGap* resultList) {
    findGaps(globalSieve, sieveStart, sieveLengthWords, 0, resultList);
}


__global__ void kernelBoth(uint32_t* globalSieve1, uint32_t* globalSieve2, uint128_t sieveStart, uint32_t sieveLengthWords,
                           uint32_t* primeList, uint32_t* rhoList, uint32_t primeCount,
                           uint32_t* smallPrimeWheel1, uint32_t* smallPrimeWheel2,
                           uint32_t* smallPrimeWheel3, uint32_t* smallPrimeWheel4,
                           uint32_t numSieveBlocks, PrimeGap* resultList) {
    if (blockIdx.x < numSieveBlocks) {
        sieveAll(globalSieve1, sieveStart, sieveLengthWords, primeList, rhoList, primeCount,
            smallPrimeWheel1, smallPrimeWheel2, smallPrimeWheel3, smallPrimeWheel4, numSieveBlocks);
    } else {
        findGaps(globalSieve2, sieveStart - ((uint128_t) sieveLengthWords)*WORD_LENGTH, sieveLengthWords, numSieveBlocks, resultList);
    }
}


void printBigNumCPU(uint128_t result) {
    // THIS WORKS!!!
    if ((uint128_t) ((uint64_t) result) == result) {
        printf("%lu", (uint64_t) result);
    } else {
        char digits[40];
        uint32_t digNum = 1;
        for (; (digNum<=40) && result; digNum++) {
            digits[40-digNum] = (char) ('0' + (result % 10));
            result /= 10;
        }
        printf("%s", digits + (41-digNum));
    }
}

uint128_t squareMod84CPU(uint128_t a, uint128_t mod) {
    uint128_t ahi = a>>42;
	uint128_t alo = a & 0x3ffffffffffL;
    return ((((a*ahi) % mod) << 42) + a*alo) % mod;
}

bool fermatTest84CPU(uint128_t n) {
    uint128_t result = 1;
    for (int bit=84; bit>=1; bit--) {
        if ((n >> bit) & 1) {
            result *= 2;
            result -= n * (result >= n);
        }
        result = squareMod84CPU(result, n);
    }
    return result == 1;
}

uint32_t modExp32CPU(uint32_t base, uint64_t n, uint32_t mod) {
    // THIS WORKS!!!
    uint64_t result = 1;
    for (int bit=32; bit>=0; bit--) {
        result = (result * result) % mod;
        if ((n >> bit) & 1) {
            result = (result * base) % mod;
        }
    }
    return (uint32_t) result;
}

void deviceInfo() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    
    printf("Number of devices: %d\n", nDevices);
    
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
                prop.memoryClockRate/1024);
        printf("  Memory Bus Width (bits): %d\n",
                prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
                2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
        printf("  Number of multiprocessors: %d\n",prop.multiProcessorCount);
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
    }
}

uint32_t* sieveInitialSmallPrimes(uint32_t limit) {
    // THIS WORKS!!!

    // here, if we just do bool[...] sieve then we can get a segfault (OOM error) for large sizes
    uint32_t* sieve = new uint32_t[limit/2];
    sieve[0] = 1;
    for (int i=1; i<limit/2; i++) {
        sieve[i] = 0;
    }
    uint32_t pr = 3;
    while (pr*pr <= limit) {
        for (int hit=pr*pr/2; hit<limit/2; hit += pr) {
            // sieve[hit] will store the lowest prime factor of hit*2+1, or 0 if there it's prime
            if (sieve[hit] == 0) sieve[hit] = pr;
        }
        do {
            pr += 2;
        } while (sieve[pr/2]);
    }
    return sieve;
}

vector<uint32_t> generateSmallPrimesList(uint32_t limit, uint32_t* sieve) {
    vector<uint32_t> primes;
    primes.push_back(2);
    for (int p=3; p<limit; p+=2) {
        if (sieve[p/2] == 0) {
            primes.push_back(p);
        }
    }
    return primes;
}

vector<uint32_t> generateRhoList(uint32_t limit, uint32_t* sieve, vector<uint32_t> primes) {
    vector<uint32_t> rhos;
    rhos.push_back(0);
    for (auto &p : primes) {
        if (p == 2) continue;
        uint32_t rho = p-1;
        while (rho%2 == 0 && modExp32CPU(2, rho/2, p) == 1) {
            rho /= 2;
        }
        uint32_t remaining = rho;
        while (remaining%2 == 0) {
            remaining /= 2;
        }
        int idx = remaining / 2;
        while (sieve[idx] > 1) {
            if (modExp32CPU(2, rho/sieve[idx], p) == 1) {
                rho /= sieve[idx];
            }
            remaining /= sieve[idx];
            idx /= sieve[idx];
        }
        if (modExp32CPU(2, rho/remaining, p) == 1) {
            rho /= remaining;
        }
        rhos.push_back(rho);
    }
    return rhos;
}

void cpuFindGapAround(uint128_t n, uint32_t minGap) {
    uint128_t p1 = n;
    p1 -= 1 - (p1 % 2);
    while (!fermatTest84CPU(p1)) {p1 -= 2;}

    uint128_t p2 = n;
    p2 += 1 - (p2 % 2);
    while (!fermatTest84CPU(p2)) {p2 += 2;}

    int gap = (int) (p2-p1);
    if (gap >= minGap) {
        printf("%lu%019lu %lu%019lu %d\n",
               hi19c(p1), lo19c(p1),
               hi19c(p2), lo19c(p2), gap);
    }
}










__global__ void sievePseudoprimesSeparate(uint128_t start, uint64_t sieveLengthWords,
                                          uint32_t* primeList, uint32_t* rhoList, uint32_t primeCount) {
    uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    for (uint32_t pidx = tidx; pidx < primeCount; pidx += stride) {
        uint32_t p = primeList[pidx];
        uint32_t rho = rhoList[pidx];
        uint64_t pTimesRho = ((uint64_t) p) * rho;
        pTimesRho <<= pTimesRho % 2; // if it's odd, multiply it by 2 - TODO: PRECOMPUTE THIS IN RHO??
        //x = ((start - p) / pTimesRho + 1) * pTimesRho - start;
        uint64_t position = pTimesRho + p - (start % pTimesRho);
        position -= pTimesRho * (position > pTimesRho);
        
        uint64_t currentWord = position / WORD_LENGTH; // this needs to be 64 bit
        uint32_t currentPosInWord = position % WORD_LENGTH;
        while (currentWord < sieveLengthWords) {
            if (IS_COPRIME_30[(currentPosInWord % 30) / 2]) {
                uint128_t num = start + ((uint128_t) currentWord)*WORD_LENGTH + currentPosInWord;
                if (num%7 && num%11 && num%13 && num%17 && num%19 && num%23 && num%29 && num%31) {
                    if (FERMAT_TEST(num)) {
                        printBigNum(num);
                        //printf("Pseudoprime p=%d mod 1e19=%lu %lu %u %u\n", p, (uint64_t) (num % 10000000000000000000UL),
                        //currentWord, currentPosInWord, (uint32_t) (num%p));
                        //printf("20000%lu\n", (uint64_t) (num % 10000000000000000000UL));
                    }
                }
            }
            

            // Find the next position
            currentPosInWord += pTimesRho % WORD_LENGTH;
            currentWord += (uint64_t) (pTimesRho / WORD_LENGTH) + (currentPosInWord >= WORD_LENGTH);
            currentPosInWord -= WORD_LENGTH * (currentPosInWord >= WORD_LENGTH);
        }
    }
}


uint128_t atouint128_t(const char *s) {
    // https://stackoverflow.com/questions/45608424/atoi-for-int128-t-type
    const char *p = s;
    uint128_t val = 0;

    while (*p >= '0' && *p <= '9') {
        val = (10 * val) + (*p - '0');
        p++;
    }
    return val;
}

void displayResultsAndClear(PrimeGap* resultList) {
    sort(resultList+1, resultList+resultList[0].gap+1, compareByPrime);
    for (int i=1; i<=resultList[0].gap; i++) {
        uint128_t endPrime = resultList[i].startPrime + resultList[i].gap;
        printf("%lu%019lu %lu%019lu %u\n",
            hi19c(resultList[i].startPrime), lo19c(resultList[i].startPrime),
            hi19c(endPrime), lo19c(endPrime),
            resultList[i].gap);
        resultList[i].startPrime = 0;
        resultList[i].gap = 0;
    }
    resultList[0].gap = 0;
}


int main(int argc, char* argv[]) {
    setbuf(stdout, NULL);
    if (argc < 2) {
        printf("Incorrect amount of command line arguments (got %d, expected 2)\n", argc);
        exit(1);
    }
    int DEVICE_NUM = 0;
    if (argc > 2) DEVICE_NUM = atoi(argv[2]);
    hipSetDevice(DEVICE_NUM);

    //deviceInfo();

    printf("Starting\n");

    int SMALL_PRIME_LIMIT = 5000000; // don't change this

    printf("Generating primes below %u\n", SMALL_PRIME_LIMIT);
    uint32_t* smallSieve = sieveInitialSmallPrimes(SMALL_PRIME_LIMIT);
    vector<uint32_t> primeList = generateSmallPrimesList(SMALL_PRIME_LIMIT, smallSieve);
    vector<uint32_t> rhoList = generateRhoList(SMALL_PRIME_LIMIT, smallSieve, primeList);
    printf("Done generating primes below %u\n", SMALL_PRIME_LIMIT);
    delete smallSieve;

    uint32_t* primeListCuda;
    auto err = hipMallocManaged(&primeListCuda, primeList.size() * sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate managed memory: %s\n", hipGetErrorString(err));
        return 1;
    }
    uint32_t* rhoListCuda;
    err = hipMallocManaged(&rhoListCuda, rhoList.size() * sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate managed memory: %s\n", hipGetErrorString(err));
        return 1;
    }
    for (int i=0; i<primeList.size(); i++) {
        primeListCuda[i] = primeList[i];
        rhoListCuda[i] = rhoList[i];
    }


    // pseudoprime is 21693774589725076147 (67 mod 120), with start 21693774589725076080
    // first kilogap above that is 21693776423220625951-6953, or 1833495550873=1.8e12 larger
    uint128_t sieveStart = atouint128_t(argv[1]);
    
    uint64_t sieveLength = BLOCK_SIZE;
    if (sieveLength >= 4294967296L * WORD_LENGTH) {
        printf("ERROR: Sieve length too large: %ld (maximum is 2^32 * %d)\n", sieveLength, WORD_LENGTH);
        exit(1);
    }
    if (sieveStart % WORD_LENGTH) {
        printf("ERROR: Sieve start must be a multiple of %d\n", WORD_LENGTH);
        exit(1);
    }
    uint64_t sieveLengthWords = sieveLength / WORD_LENGTH;

    /*int offset=78498; // doing primes from 1M to 1.1M
    printf("using asdf %u\n", (primeList.size()-offset));
    
    auto startp = chrono::high_resolution_clock::now();
    sievePseudoprimesSeparate<<<384,64>>>(sieveStart, 100000000000000UL,
                                          primeListCuda+offset, rhoListCuda+offset, 7216); //primeList.size()-offset);
    hipDeviceSynchronize();
    auto finishp = chrono::high_resolution_clock::now();
    cout << "Done in " << chrono::duration_cast<chrono::nanoseconds>(finishp-startp).count()/1e9 << " seconds\n";
    return 0;*/
    
    uint128_t* endpoints;
    hipMalloc((void **) &endpoints, sieveLengthWords/12288 * 2 * sizeof(uint128_t));




    uint32_t* smallPrimeWheel1;
    uint32_t* smallPrimeWheel2;
    uint32_t* smallPrimeWheel3;
    uint32_t* smallPrimeWheel4;
    hipMalloc((void **) &smallPrimeWheel1, (7*11*13*17*19*23*29) * sizeof(uint32_t));
    hipMalloc((void **) &smallPrimeWheel2, (31*37*41*43*47) * sizeof(uint32_t));
    hipMalloc((void **) &smallPrimeWheel3, (53*59*61*67) * sizeof(uint32_t));
    hipMalloc((void **) &smallPrimeWheel4, (71*73*79*83) * sizeof(uint32_t));
    printf("Making small prime sieve\n");
    
    auto start1 = chrono::high_resolution_clock::now();
    makeSmallPrimeWheels<<<96,512>>>(smallPrimeWheel1, smallPrimeWheel2, smallPrimeWheel3, smallPrimeWheel4);
    auto finish1 = chrono::high_resolution_clock::now();
    cout << "Done in " << chrono::duration_cast<chrono::nanoseconds>(finish1-start1).count()/1e9 << " seconds\n";
   
    PrimeGap* resultList;
    err = hipMallocManaged((void **) &resultList, sizeof(PrimeGap) * RESULT_LIST_SIZE);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate managed memory: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    uint32_t* globalSieve1;
    uint32_t* globalSieve2;
    hipMalloc((void **) &globalSieve1, sieveLengthWords * sizeof(uint32_t));
    hipMalloc((void **) &globalSieve2, sieveLengthWords * sizeof(uint32_t));


    hipMemset(globalSieve1, 0, sieveLengthWords * sizeof(uint32_t));
    kernel<<<96,512>>>(
        globalSieve1, sieveStart, (uint32_t) sieveLengthWords,
        primeListCuda, rhoListCuda, primeList.size(),
        smallPrimeWheel1, smallPrimeWheel2, smallPrimeWheel3, smallPrimeWheel4
    );
    hipDeviceSynchronize();

    for (int i=0; i<RESULT_LIST_SIZE; i++) {
        resultList[i].startPrime = 0;
        resultList[i].gap = 0;
    }

    if (MIN_GAP_SIZE % 30) {
        printf("Searching for gaps of size >= %d... (modified from %d)\n", MIN_GAP_SIZE - (MIN_GAP_SIZE % 30), MIN_GAP_SIZE);
    } else {
        printf("Searching for gaps of size >= %d...\n", MIN_GAP_SIZE);
    }

    int blocksToTest = 100;
    auto start = chrono::high_resolution_clock::now();
    auto finish = start;
    for (int i=0; i<blocksToTest-1; i++) {
        if (i%PROGRESS_UPDATE_BLOCKS == 0) {
            finish = chrono::high_resolution_clock::now();
            printf("Done %d blocks (limit=%lu%019lu time=%f seconds)\n", i, hi19c(sieveStart), lo19c(sieveStart),
                    chrono::duration_cast<chrono::nanoseconds>(finish-start).count()/1e9);
        }
        hipMemset(globalSieve2, 0, sieveLengthWords * sizeof(uint32_t));
        
        kernelBoth<<<192,512>>>(
            globalSieve2, globalSieve1, sieveStart+sieveLength, (uint32_t) sieveLengthWords,
            primeListCuda, rhoListCuda, primeList.size(),
            smallPrimeWheel1, smallPrimeWheel2, smallPrimeWheel3, smallPrimeWheel4, 120,
            resultList
        );
        cpuFindGapAround(sieveStart, MIN_GAP_SIZE - (MIN_GAP_SIZE % 30)); // TODO: ADD THIS TO THE LIST IF WE FIND ONE!!!
        hipDeviceSynchronize();
        displayResultsAndClear(resultList);
        sieveStart += sieveLength;
        swap(globalSieve1, globalSieve2);
    }

    if ((blocksToTest-1)%PROGRESS_UPDATE_BLOCKS == 0) {
        finish = chrono::high_resolution_clock::now();
        printf("Done %d blocks (limit=%lu%019lu time=%f seconds)\n", blocksToTest-1, hi19c(sieveStart), lo19c(sieveStart),
            chrono::duration_cast<chrono::nanoseconds>(finish-start).count()/1e9);
    }

    for (int i=0; i<RESULT_LIST_SIZE; i++) {
        resultList[i].startPrime = 0;
        resultList[i].gap = 0;
    }
    kernel2<<<96,256>>>(globalSieve1, sieveStart, sieveLengthWords, resultList);
    cpuFindGapAround(sieveStart, MIN_GAP_SIZE - (MIN_GAP_SIZE % 30));
    hipDeviceSynchronize();
    displayResultsAndClear(resultList);
    sieveStart += sieveLength;

    finish = chrono::high_resolution_clock::now();
    printf("Done %d blocks (limit=%lu%019lu time=%f seconds)\n", blocksToTest, hi19c(sieveStart), lo19c(sieveStart),
        chrono::duration_cast<chrono::nanoseconds>(finish-start).count()/1e9);

    return 0;
}

/*
- General tips:
    - Don't be afraid to UNROLL LOOPS!
    - Remember to __syncthreads() between each step!
    - We need to use an AtomicOr function on ALL STEPS!!! Because some blocks might be in a different step than others!
        - This includes the shared memory step, because we cannot miss any bits with sieving small primes due to PSPs
    - Maximum amount of shared memory per block is 49152 bytes, or 12288 ints
    - Maximum amount of constant memory is 65536 bytes, or 16384 ints

- General structure of the code:
    - We have 2 kernels, one for setting up the global memory sieve and the other for iterating over it
    - The CPU code is essentially (for each block of 36 billion) (do kernel 1 then kernel 2)
    - POTENTIAL PROBLEM WITH 2 KERNELS: If we separate the global memory accesses from the fermat tests, then
        it could be less efficient than if we did them separately (we have to parallelize fermat tests and global memory)

- We have a global(?) variable that signifies what multiple of 2^64 we are using
    - we have to precompute (2^64*that) % p for primes p

- The sieve is a pointer to ints with another int that signifies its size
- Each int has 32 bits, which covers 120 numbers because we are skipping multiples of 2,3,5 (the tiny primes)
    - We will probably need constant arrays to help with this format
- We have constant memory that has wheels for small primes starting from 7 up to let's say 47.

- Next step is medium primes, we use shared memory for this.
    - Parallelize over the list of primes
    - Can't use buckets here, too much memory.
    - So we will have to calculate modulos. But we can do this with fastmod by precomputing inverses for all primes.
    - When we calculate the offset, we need to worry about the 120/32 format of the bits

- Next is big primes, we use global memory for this.
    https://github.com/kimwalisch/primesieve/blob/master/doc/ALGORITHMS.md
    - Use a bucket sieve:
    - List of lists [L1, L2, L3, L4...] where the nth list contains all primes whose next multiple is in that subinterval
    - we use wraparound for larger subintervals while we sieve
    - the reason for this is we don't want to kill our memory usage
    - if the shared memory list length is 1024 ints, that can hold 1024*120 = 122,880 numbers

- Next step is PSP sieving.
    - For any remaining primes up to 5 million or whatever our precomputed PSP limit is,
    - Precompute all rho(p) values, parallelize over the list of primes
    - Remove values that are p (mod p*rho(p))

- The final step is fermat tests.
    - Global memory access is negligible here, since the actual fermat tests take longer
    - If we find a prime, skip forward by the minimum gap size, and search backward for a prime
    - If we get back to where we started without finding one, go back but search forward instead, then print that gap
    - Be smart with this code so that we can fully parallelize the fermat tests


THINGS TO ADD:

settings/worktodo files: (should put this in the readme at some point)

=== settings.txt file format: ===
# comment: there should be a script to automatically find the optimal parameters to set
GPU_BLOCKS=192
NUM_BLOCKS_FOR_SIEVING=120
GPU_THREADS=512
BLOCK_SIZE=46080000000
SORT_OUTPUT_BY_GAPSIZE=1 # If 0, sorts by the prime (increasing). If 1, sorts by the gap size (decreasing)
NAME=B.Kehrig

=== worktodo.txt file format: ===
# format: start(e12), end(e12), minGap, username
18470057,18571674,1200,B.Kehrig  #will find the 1552 and 1572 gaps, but also will take a while (smaller tasks recommended)
# worktodo will have an ETA

=== output file format: === (location: output/gaps_<start>e12_<end>e12_min<mingap>_<name>.txt)
===== PRIME GAP REPORT =====
Minimum gap size: <mingap>
Gaps >=1200: <x> (or whatever hundred is at least as large as mingap)
Gaps >=1300: <x>
... keep going until there are none left
Largest gap: <size> <prime>

All gaps >= <mingap>: # format: <gapsize> <startprime> <merit> <name>
1572 18571673432051830099 35.430806 B.Kehrig
1552 18470057946260698231 34.984359 B.Kehrig # (these would be in the opposite order if SORT_OUTPUPT_BY_GAPSIZE=0)


How do I write TESTS???
Use a compile-time variable RUN_TESTS
if set, it will do a run with some specified settings (start num, block size, etc...) to stay consistent
it will look at certain values from sieving and pseudoprime sieving and compare that to the correct values
For testing the primality tests, ???

I'm pretty sure that for the 4090, the bottleneck is MEMORY ACCESSES, so it cannot benefit from any more fermat test upgrades.
For my laptop GPU, speeding up the Fermat can still help.
*/